#include "hip/hip_runtime.h"
#include <model/PillarsBuilderCuda.h>

#include <Eigen/Core>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

__device__ bool IsInRoi(float x, float y)
{
    return x > Config::roi_x_min && x < Config::roi_x_max && y > Config::roi_y_min && y < Config::roi_y_max;
}

__device__ int Point2Index(float x, float y)
{
    return (x - Config::roi_x_min) / Config::pillar_x_size * Config::bev_h +
           (y - Config::roi_y_min) / Config::pillar_y_size;
};

__device__ Eigen::Vector2i Point2Index2d(float x, float y)
{
    return Eigen::Vector2i((x - Config::roi_x_min) / Config::pillar_x_size,
                           (y - Config::roi_y_min) / Config::pillar_y_size);
};

class PillarsBuilderCuda
{
public:
    struct PillarInfo
    {
        Eigen::Vector3f mean; // 记录pillar中心点坐标
        int num;              // 记录pillar中的有效点数目
        int idx;              // 记录pillar对应在稠密张量中的索引
    };

private:
    thrust::device_vector<Eigen::Matrix<float, 1, 8>> raw_cloud_; // 原始点云 按照PCL XYZI格式的设置为8个float

    const int pillars_nums_ = Config::bev_h * Config::bev_w; // pill的数目
    thrust::device_vector<PillarInfo> pillars_;

    thrust::device_vector<Eigen::Vector2i> pillar_idx_;
    thrust::device_vector<Eigen::Matrix<float, 1, 9>> pillar_feature_;

public:
    PillarsBuilderCuda();
    void BuildPillarsFeature(const float *data, int point_num);
    auto &GetPillarIndex() { return pillar_idx_; }
    auto &GetPillarFeatureData() { return pillar_feature_; }
};

PillarsBuilderCuda::PillarsBuilderCuda()
{
    pillars_.resize(pillars_nums_);
};

void PillarsBuilderCuda::BuildPillarsFeature(const float *data, int point_num)
{
    // 1. 数据传输到 GPU
    raw_cloud_.resize(point_num);
    auto data_reint = reinterpret_cast<const Eigen::Matrix<float, 1, 8> *>(data);
    thrust::copy(data_reint, data_reint + point_num, raw_cloud_.begin());

    // 2. 重置所有Pillar的数据
    thrust::for_each_n(pillars_.begin(), pillars_nums_,
                       [] __device__(PillarInfo & pillar)
                       {
                           pillar.mean = Eigen::Vector3f::Zero();
                           pillar.num = 0;
                       });

    // 3. 计算每个Pillar的sum和num

    auto pillars_ptr = thrust::raw_pointer_cast(pillars_.data());
    thrust::for_each_n(raw_cloud_.begin(), point_num,
                       [=] __device__(auto &point)
                       {
                           if (IsInRoi(point[0], point[1]))
                           {
                               int index = Point2Index(point[0], point[1]);
                               atomicAdd(&pillars_ptr[index].num, 1);
                               atomicAdd(&pillars_ptr[index].mean.x(), point[0]);
                               atomicAdd(&pillars_ptr[index].mean.y(), point[1]);
                               atomicAdd(&pillars_ptr[index].mean.z(), point[2]);
                           }
                       });

    // 4. 计算每个Pillar的mean并得到每个有效pillar在稠密张量中对应的索引
    thrust::device_vector<int> valid_pillars(1, 0);
    auto valid_pillars_ptr = thrust::raw_pointer_cast(valid_pillars.data());
    thrust::for_each_n(pillars_.begin(), pillars_nums_,
                       [=] __device__(PillarInfo & pillar)
                       {
                           if (pillar.num > 0)
                           {
                               pillar.mean = pillar.mean / pillar.num;
                               pillar.idx = atomicAdd(valid_pillars_ptr, 1);
                               pillar.num = 0; // 这里是为了下面重新用这个量记录有效的点数量
                           }
                           else
                           {
                               pillar.idx = -1;
                           }
                       });

    // 5. 构建稠密pillar特征
    int pillar_num = valid_pillars[0];
    pillar_feature_.resize(pillar_num * Config::max_nums_in_pillar, Eigen::Matrix<float, 1, 9>::Zero());
    pillar_idx_.resize(pillar_num, Eigen::Vector2i::Zero());
    auto pillar_feature_ptr = thrust::raw_pointer_cast(pillar_feature_.data());
    auto pillar_idx_ptr = thrust::raw_pointer_cast(pillar_idx_.data());
    thrust::for_each_n(raw_cloud_.begin(), point_num,
                       [=] __device__(auto &point)
                       {
                           if (IsInRoi(point[0], point[1]))
                           {
                               int index = Point2Index(point[0], point[1]);
                               int idx_curr = atomicAdd(&pillars_ptr[index].num, 1);
                               if (idx_curr < Config::max_nums_in_pillar)
                               {
                                   int idx_feature = pillars_ptr[index].idx * Config::max_nums_in_pillar + idx_curr;
                                   auto index2d = Point2Index2d(point[0], point[1]);

                                   pillar_feature_ptr[idx_feature] << index2d.x(), index2d.y(), point[0], point[1],
                                       point[2], point[4], point[0] - pillars_ptr[index].mean.x(),
                                       point[1] - pillars_ptr[index].mean.y(), point[2] - pillars_ptr[index].mean.z();
                                   if (idx_curr == 0)
                                   {
                                       pillar_idx_ptr[pillars_ptr[index].idx] << index2d.x(), index2d.y();
                                   }
                               }
                           }
                       });
}

std::tuple<float *, int *, int> BuildPillarsFeature(const float *data, int point_num)
{
    static PillarsBuilderCuda pillars_builder_cuda;
    pillars_builder_cuda.BuildPillarsFeature(data, point_num);
    return std::make_tuple(
        reinterpret_cast<float *>(thrust::raw_pointer_cast(pillars_builder_cuda.GetPillarFeatureData().data())),
        reinterpret_cast<int *>(thrust::raw_pointer_cast(pillars_builder_cuda.GetPillarIndex().data())),
        pillars_builder_cuda.GetPillarIndex().size());
}
